#include <gpu_monitor.h>
#include <ros_gpu_monitor/GPUProperties.h>
#include <ros_gpu_monitor/CudaInfo.h>
#include <ros_gpu_monitor/BytesWithUnit.h>
#include <ros_gpu_monitor/HzWithUnit.h>

//headers for cuda
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

gpu_monitor::gpu_monitor()
{
  int device_id;
  nh_.param<int>(ros::this_node::getName()+"/device_id", device_id, 0);
  nh_.param<int>(ros::this_node::getName()+"/publish_rate", publish_rate_, 10);
  get_device_info_(device_id);
  gpu_properties_pub = nh_.advertise<ros_gpu_monitor::GpuProperties>("gpu_properties", 1);
}

gpu_monitor::~gpu_monitor()
{

}

void gpu_monitor::run()
{
  ros::Rate rate = ros::Rate(publish_rate_);
  while (ros::ok())
  {
    gpu_properties_pub.publish(gpu_properties_msg_);
    rate.sleep();
  }
}

bool gpu_monitor::get_device_info_(int device_id)
{
  hipSetDevice(device_id);
  hipError_t error_info = hipGetDeviceProperties(&device_properties_, device_id);
  if(error_info != hipSuccess)
  {
    ROS_WARN_STREAM(hipGetErrorString(error_info));
    return false;
  }
  ROS_INFO_STREAM("cuda device type = " << device_properties_.name);
  gpu_properties_msg_.gpu_type = device_properties_.name;
  gpu_properties_msg_.device_id = device_id;
  int runtime_version = 0;
  int driver_version = 0;
  hipRuntimeGetVersion(&runtime_version);
  hipDriverGetVersion(&driver_version);
  gpu_properties_msg_.cuda_info.runtime_version = (float)runtime_version/1000;
  gpu_properties_msg_.cuda_info.driver_version = (float)driver_version/1000;
  gpu_properties_msg_.global_memory.bytes = (float)device_properties_.totalGlobalMem/1024/1024;
  gpu_properties_msg_.global_memory.data_unit = gpu_properties_msg_.global_memory.MB;
  gpu_properties_msg_.multiprocessors = device_properties_.multiProcessorCount;
  //gpu_properties_msg_.cuda_cores = _ConvertSMVer2Cores(device_properties_.major, device_properties_.minor) * device_properties_.multiProcessorCount);
  gpu_properties_msg_.gpu_max_clock_rate.hz = device_properties_.clockRate*1e-6f;
  gpu_properties_msg_.gpu_max_clock_rate.data_unit = gpu_properties_msg_.gpu_max_clock_rate.GHZ;
  gpu_properties_msg_.shared_memory_per_block.bytes = (float)device_properties_.sharedMemPerBlock/1024;
  gpu_properties_msg_.shared_memory_per_block.data_unit = gpu_properties_msg_.shared_memory_per_block.KB;
  gpu_properties_msg_.register_per_block = device_properties_.regsPerBlock;
  gpu_properties_msg_.warp_size = device_properties_.warpSize;
  gpu_properties_msg_.max_pitch.bytes = (float)device_properties_.memPitch;
  gpu_properties_msg_.max_pitch.data_unit = gpu_properties_msg_.max_pitch.BYTE;
  gpu_properties_msg_.max_size_of_each_dimension_of_block[0] = device_properties_.maxThreadsDim[0];
  gpu_properties_msg_.max_size_of_each_dimension_of_block[1] = device_properties_.maxThreadsDim[1];
  gpu_properties_msg_.max_size_of_each_dimension_of_block[2] = device_properties_.maxThreadsDim[2];
  gpu_properties_msg_.max_size_of_each_dimension_of_grid[0] = device_properties_.maxGridSize[0];
  gpu_properties_msg_.max_size_of_each_dimension_of_grid[1] = device_properties_.maxGridSize[1];
  gpu_properties_msg_.max_size_of_each_dimension_of_grid[2] = device_properties_.maxGridSize[2];
  gpu_properties_msg_.constant_memory.bytes = (float)device_properties_.totalConstMem/1024;
  gpu_properties_msg_.constant_memory.data_unit = gpu_properties_msg_.constant_memory.KB;
  gpu_properties_msg_.cuda_info.capability_major_version = device_properties_.major;
  gpu_properties_msg_.cuda_info.capability_minor_version = device_properties_.minor;
  return true;
}
